#include "hip/hip_runtime.h"
#ifndef SERIAL



#include "config.h"
#include "timer.h"
#include "vec_oper.h"
#include "k_vec_oper.h"

#include <stdint.h>
#include <stddef.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>



time_s Operation(arr_t* arr1, arr_t* arr2, arr_t* out, const uint32_t& size, const Oper& op) {
  time_s time;
  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);

  arr_t *d_arr1, *d_arr2, *d_out;
  uint32_t *d_size;
  hipMalloc((void**)&d_arr1, size*sizeof(arr_t));
  hipMalloc((void**)&d_arr2, size*sizeof(arr_t));
  hipMalloc((void**)&d_out, size*sizeof(arr_t));
  hipMalloc((void**)&d_size, sizeof(uint32_t));

  hipStream_t stream;
  hipStreamCreate(&stream);


  CUDATIME(({
    hipHostRegister(arr1, size*sizeof(arr_t), hipHostRegisterDefault);
    hipHostRegister(arr2, size*sizeof(arr_t), hipHostRegisterDefault);

    hipMemcpyAsync(d_arr1, arr1, size*sizeof(arr_t), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_arr2, arr2, size*sizeof(arr_t), hipMemcpyHostToDevice, stream);
    hipMemcpy(d_size, &size, sizeof(uint32_t), hipMemcpyHostToDevice);

    hipHostUnregister(arr1);
    hipHostUnregister(arr2);
  }), time.memcpy, start, end);

  dim3 blocks(KBLOCKS, 1, 1);
  dim3 threads(KTHREADS, 1, 1);

  CUDATIME(({
    switch(op) {
      case opadd : KAdd<<<blocks,threads>>>(d_arr1, d_arr2, d_out, *d_size); break;
      case opsub : KSub<<<blocks,threads>>>(d_arr1, d_arr2, d_out, *d_size); break;
      case opmul : KMul<<<blocks,threads>>>(d_arr1, d_arr2, d_out, *d_size); break;
      case opdiv : KDiv<<<blocks,threads>>>(d_arr1, d_arr2, d_out, *d_size); break;
      default : break;
    } 
  }), time.run, start, end);

  CUDATIME(({
    hipHostRegister(out, size*sizeof(arr_t), hipHostRegisterDefault);

    hipMemcpyAsync(out, d_out, size*sizeof(arr_t), hipMemcpyDeviceToHost, stream);

    hipHostUnregister(out);
  }), time.memret, start, end); 

  time.total = time.memcpy + time.run + time.memret;

  hipStreamSynchronize(stream);
  hipStreamDestroy(stream);

  hipFree(d_arr1);
  hipFree(d_arr2);
  hipFree(d_out);
  hipFree(d_size);

  return time;
}



__global__ void KAdd(arr_t* arr1, arr_t* arr2, arr_t* out, const uint32_t& size) {
  uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < size)
    out[idx] = arr1[idx] + arr2[idx];
}

__global__ void KSub(arr_t* arr1, arr_t* arr2, arr_t* out, const uint32_t& size) {
  uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < size)
    out[idx] = arr1[idx] - arr2[idx];
}

__global__ void KMul(arr_t* arr1, arr_t* arr2, arr_t* out, const uint32_t& size) {
  uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < size)
    out[idx] = arr1[idx] * arr2[idx];
}

__global__ void KDiv(arr_t* arr1, arr_t* arr2, arr_t* out, const uint32_t& size) {
  uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < size && arr2[idx])
    out[idx] = arr1[idx] / arr2[idx];
}

#endif
